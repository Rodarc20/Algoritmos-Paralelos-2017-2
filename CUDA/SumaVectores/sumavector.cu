
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string>

__global__
void vecAddKernel(float * a, float * b, float * c, int n){//funcion ejecutada en la gpu, osea la funcion device
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
        *(c+i) = *(a+i) + *(b+i);//c[i] = a[i] + b[i]
}

void vecAdd(float * a, float * b, float * c, int n){
    int size = n * sizeof(float);//calcular el tamaño de memoria de los vectores
    float * d_a, * d_b, * d_c;
    hipMalloc((void **) & d_a, size);//creacion de los vectores en la memoria del gpu. //es necesario el void **?
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);//destino, origen, tamaño, 
    hipMalloc((void **) & d_b, size);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMalloc((void **) & d_c, size);

    //llamada al la funcon kernel con ciertos parametros
    //<<<numero de bloques, tamaño de los bloques>>>, el numero de bloques es el techo de (tamaño de elementos / tamaño del bloque), el tamaño del bloque es el numero de threads del bloque
    //el numero recibido por la funcion ceil (techo) debe ser un flotante, por eso el .0
    vecAddKernel<<<ceil(n/256.0), 256>>> (d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    //liberando memoria del device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

int main(){
    int n = 1 << 20;//probar con 20
    float * a, * b, * c;
    a = (float *)malloc(n*sizeof(float));//los malloc fevuelven un tipo void * ahy que convertirlo a lfoat *
    b = (float *)malloc(n*sizeof(float));
    c = (float *)malloc(n*sizeof(float));

    for(int i = 0; i < n; i++){
        *(a+i) = i;
        *(b+i) = i;
    }
    vecAdd(a, b, c, n);
    /*for(int i = 0; i < n; i++){
        printf("%f ", *(c+i));
    }*/
    printf("Finalizado\n");
    return 0;
}
